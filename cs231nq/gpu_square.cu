#include "hip/hip_runtime.h"
// Include the cuda header and the k.h interface.
#include </usr/local/cuda/include/hip/hip_runtime.h>
#include"stdio.h"
#include"k.h"

// Export the function we will load into kdb+
extern  "C" K gpu_square(K x);

// Define the "Kernel" that executes on the CUDA device in parallel
__global__ void square_array(double *a, int N) {
 int idx = blockIdx.x * blockDim.x + threadIdx.x;
 if (idx<N)
    a[idx] = a[idx] * a[idx];
}

// A function to use from kdb+ to square a vector of reals by
// - allocating space on the graphics card
// - copying the data over from the K object
// - doing the work
// - copy back and overwrite the K object data
K gpu_square(K x) {
  // Pointers to host & device arrays
 double *host_memory = (float*) &(kF(x)[0]), *device_memory;

 // Allocate memory on the device for the data and copy it to the GPU
 size_t size = xn * sizeof(double);
 hipMalloc((void **)&device_memory, size);
 hipMemcpy(device_memory, host_memory, size, hipMemcpyHostToDevice);

 // Do the computaton on the card
 int block_size = 4;
 int n_blocks = xn/block_size + (xn%block_size == 0 ? 0:1);
 square_array <<< n_blocks, block_size >>> (device_memory, xn);

 // Copy back the data, overwriting the input, 
 // free the memory we allocated on the graphics card
 hipMemcpy(host_memory, device_memory, size, hipMemcpyDeviceToHost);
 hipFree(device_memory);
 R r1(x);
}

