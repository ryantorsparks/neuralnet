#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<unistd.h>
#include"k.h"
#include<math.h>

// CUDA Headers
#include </usr/local/cuda/include/hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""

// Helper definition
#define VAR(v, i) __typeof(i) v=(i)
#define FOR(i, j, k) for (int i = (j); i <= (k); ++i)
#define REP(i, n) for(int i = 0;i <(n); ++i)

// CONSTS
#define INF     1061109567 // 3F 3F 3F 3F
#define CHARINF 63       // 3F    
#define CHARBIT 8
#define NONE    -1

#define CMCPYHTD hipMemcpyHostToDevice
#define CMCPYDTH hipMemcpyDeviceToHost

// CONSTS for compute capability 2.0
#define BLOCK_WIDTH 16
#define WARP         32

/** Cuda handle error, if err is not success print error and line in code
*
* @param status CUDA Error types
*/
#define HANDLE_ERROR(err) \
{ \
    if (err != hipSuccess) \
    { \
        fprintf(stderr, "%s failed  at line %d \nError message: %s \n", \
            __FILE__, __LINE__ ,hipGetErrorString(err)); \
        exit(EXIT_FAILURE); \
    } \
}

extern  "C" K gpu_floydwarshall(K matrix);

/**Kernel for wake gpu
*
* @param reps dummy variable only to perform some action
*/
__global__ void wake_gpu_kernel(int reps) 
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= reps) return;
}

/**Kernel for parallel Floyd Warshall algorithm on gpu
* 
* @param u number vertex of which is performed relaxation paths [v1, v2]
* @param n number of vertices in the graph G:=(V,E), n := |V(G)|
* @param d matrix of shortest paths d(G)
*/
__global__ void fw_kernel(const unsigned int u, const unsigned int n, int * const d)
{
    int v1 = blockDim.y * blockIdx.y + threadIdx.y;
    int v2 = blockDim.x * blockIdx.x + threadIdx.x;
    
    if (v1 < n && v2 < n) 
    {
        int newPath = d[v1 * n + u] + d[u * n + v2];
        int oldPath = d[v1 * n + v2];
        if (oldPath > newPath)
        {
            d[v1 * n + v2] = newPath;
        }
    }
}

K gpu_floydwarshall(K matrix)
{
    unsigned int V = sqrt(matrix->n);
    unsigned int n = V;
    // Alloc host data for G - graph, d - matrix of shortest paths
    unsigned int size = V * V;
    int *d = (int *) malloc (sizeof(int) * size);
    int *dev_d = 0;
    hipError_t cudaStatus;
    hipStream_t cpyStream;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    HANDLE_ERROR(cudaStatus);

    // Initialize the grid and block dimensions here
    dim3 dimGrid((n - 1) / BLOCK_WIDTH + 1, (n - 1) / BLOCK_WIDTH + 1, 1);
    dim3 dimBlock(BLOCK_WIDTH, BLOCK_WIDTH, 1);

    // Create new stream to copy data
    cudaStatus = hipStreamCreate(&cpyStream);
    HANDLE_ERROR(cudaStatus);

    // Allocate GPU buffers for matrix of shortest paths d)
    cudaStatus =  hipMalloc((void**)&dev_d, n * n * sizeof(int));
    HANDLE_ERROR(cudaStatus);
 
    // Wake up gpu
    wake_gpu_kernel<<<1, dimBlock>>>(32);

    // Copy input from host memory to GPU buffers.
    int *host_memoryd = (int*)&(kI(matrix)[0]);
    cudaStatus = hipMemcpyAsync(dev_d, host_memoryd, n * n * sizeof(int), CMCPYHTD, cpyStream);

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    cudaStatus = hipDeviceSynchronize();
    HANDLE_ERROR(cudaStatus);

    hipFuncSetCacheConfig(reinterpret_cast<const void*>(fw_kernel), hipFuncCachePreferL1 );
    FOR(u, 0, n - 1)
    {
        fw_kernel<<<dimGrid, dimBlock>>>(u, n, dev_d);
    }

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    HANDLE_ERROR(cudaStatus);

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    HANDLE_ERROR(cudaStatus);

    cudaStatus = hipMemcpy(host_memoryd, dev_d, n * n * sizeof(int), CMCPYDTH);
    HANDLE_ERROR(cudaStatus);

    cudaStatus = hipFree(dev_d);
    HANDLE_ERROR(cudaStatus);

    // Delete allocated memory 
    free(d);
    R r1(matrix);
}
