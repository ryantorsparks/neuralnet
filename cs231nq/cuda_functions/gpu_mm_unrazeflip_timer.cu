#include </usr/local/cuda/include/hip/hip_runtime.h>
#include"stdio.h"
#include"k.h"
#include <iostream>
#include <cstdlib>
#include <ctime>
#include <hipblas.h>
#include <hiprand.h>
#include <time.h>
#include <sys/time.h>
#define uS_PER_SEC 1000000
#define uS_PER_mS 1000
#define N  1000
#define M 1000

// compile with:
// >> nvcc --compiler-options '-fPIC -DKXVER=3 -O2' -o $QHOME/l64/gpu_mm_unrazeflip_timer.so --shared -lcurand -lcublas gpu_mm_unrazeflip_timer.cu
// load into q with:
// q).gpu.mm:`gpu_mm_unrazeflip 2:(`gpu_mm;7)
// q).gpu.unrazeflip:`gpu_mm_unrazeflip 2:(`unrazeflip;3)
// q).gpu.mmu:{[x;y] .gpu.unrazeflip[;rows_x;cols_y] .gpu.mm[raze x;rows_x;count x 0;raze y;count y;cols_y;((rows_x:count x)*cols_y:count y 0)#0f]}
// .gpu.mm[a;b]~mmu[a;b]

// Export the function we will load into kdb+
extern  "C" K gpu_mm(K A, K rA, K cA, K B, K rB, K cB, K C);
extern  "C" K unrazeflip(K x, K rows_x, K cols_x);

// Multiply the arrays A and B on GPU and save the result in C
// C(m,n) = A(m,k) * B(k,n)
// m= nr_rows_A
// k= nr_cols_A
// n= nr_cols_B
void gpu_blas_mmul(const double *A, const double *B, double *C, const int m, const int k, const int n) {
    int lda=k,ldb=n,ldc=m;
    timeval t1, t2;
    const double alf = 1;
    const double bet = 0;
    const double *alpha = &alf;
    const double *beta = &bet;

    // Create a handle for CUBLAS
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    gettimeofday(&t1, NULL);
    // Do the actual multiplication
    // HIPBLAS_OP_T means input is row major, HIPBLAS_OP_N means input is column major
    hipblasDgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_T, m, n, k, alpha, A, lda, B, ldb, beta, C, ldc);
    gettimeofday(&t2, NULL);
    float et2 = (((t2.tv_sec*uS_PER_SEC)+t2.tv_usec) - ((t1.tv_sec*uS_PER_SEC)+t1.tv_usec))/(float)uS_PER_mS;
    printf("time to perform cublas matrix multiply: %fms\n", et2);

    // Destroy the handle
    hipblasDestroy(handle);
}

K gpu_mm(K A, K rA, K cA, K B, K rB, K cB,  K C) {
    // Allocate 3 arrays on CPU
    int nr_rows_A = rA->n;
    int nr_cols_A = cA->n;
    int nr_rows_B = rB->n;
    int nr_cols_B = cB->n;
    int nr_rows_C = nr_rows_A;
    int nr_cols_C = nr_cols_B;
    timeval t1, t2, t3, t4, t5, t6;

    // allocate memory, host arrays
        gettimeofday(&t1, NULL);
    double *h_A = (double *)malloc(nr_rows_A * nr_cols_A * sizeof(double));
    double *h_B = (double *)malloc(nr_rows_B * nr_cols_B * sizeof(double));
    double *h_C = (double *)malloc(nr_rows_C * nr_cols_C * sizeof(double));

    // Allocate 3 arrays on GPU, device arrays
    double *d_A, *d_B, *d_C;
    double *host_memoryA = (double*) &(kF(A)[0]);
    double *host_memoryB = (double*) &(kF(B)[0]);
    double *host_memoryC = (double*) &(kF(C)[0]);
    size_t sizeA = nr_rows_A * nr_cols_A * sizeof(double);
    size_t sizeB = nr_rows_B * nr_cols_B * sizeof(double);
    hipMalloc((void **)&d_A, sizeA);
    hipMalloc((void **)&d_B, sizeB);
    hipMalloc(&d_C,nr_rows_C * nr_cols_C * sizeof(double));
        gettimeofday(&t2, NULL);
        float et2 = (((t2.tv_sec*uS_PER_SEC)+t2.tv_usec) - ((t1.tv_sec*uS_PER_SEC)+t1.tv_usec))/(float)uS_PER_mS;
        printf("time to allocate host and device array mems: %fms\n", et2);


    // copy A and B to GPU:
        gettimeofday(&t3, NULL);
    hipMemcpy(d_A, host_memoryA, sizeA, hipMemcpyHostToDevice);
    hipMemcpy(d_B, host_memoryB, sizeB, hipMemcpyHostToDevice);
        gettimeofday(&t4, NULL);
        float et4 = (((t4.tv_sec*uS_PER_SEC)+t4.tv_usec) - ((t3.tv_sec*uS_PER_SEC)+t3.tv_usec))/(float)uS_PER_mS;
        printf("time to copy inputs to GPU: %fms\n", et4);

    // Multiply A and B on GPU
    gpu_blas_mmul(d_A, d_B, d_C, nr_rows_A, nr_cols_A, nr_cols_B);

    // Copy the result back to host memory
        gettimeofday(&t5, NULL);
    hipMemcpy(host_memoryC,d_C,nr_rows_C * nr_cols_C * sizeof(double),hipMemcpyDeviceToHost);
        gettimeofday(&t6, NULL);
        float et6 = (((t6.tv_sec*uS_PER_SEC)+t6.tv_usec) - ((t5.tv_sec*uS_PER_SEC)+t5.tv_usec))/(float)uS_PER_mS;
        printf("time to copy result from GPU back to host: %fms\n", et6);

    //Free GPU memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);    

    // Free CPU memory
    free(h_A);
    free(h_B);
    free(h_C);

    R r1(C);
}


// hack function, cublas returns a "razed matrix transpose", i.e. .gpu.mm[x;y]~raze flip mmu[x;y]
// so this basically undoes this
K unrazeflip(K x, K rows_x, K cols_x){
    long r=rows_x->n;
    long c=cols_x->n;
    K res, row;
    res = ktn(0,0);
    long j=0;
    long i=0;
    for(j=0;j<r;++j){
       row = ktn(KF,c);
       for(i=0;i<c;++i){
          kF(row)[i]=kF(x)[(i*r)+j];
       }
       jk(&res,row);
    }
    return res;
}
