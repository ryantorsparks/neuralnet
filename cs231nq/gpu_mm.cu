#include "hip/hip_runtime.h"
#include </usr/local/cuda/include/hip/hip_runtime.h>
#include"stdio.h"
#include"k.h"
#include <iostream>
#include <cstdlib>
#include <ctime>
#include <hipblas.h>
#include <hiprand.h>

// compile with:
//   nvcc --compiler-options '-fPIC -DKXVER=3 -O2' -o $QHOME/l64/gpu_mm.so --shared -lcurand -lcublas gpu_mm.cu

// Export the function we will load into kdb+
extern  "C" K gpu_mm(K A, K B, K C);

// Fill the array A(nr_rows_A, nr_cols_A) with random numbers on GPU
void GPU_fill_rand(float *A, int nr_rows_A, int nr_cols_A) {
    // Create a pseudo-random number generator
    hiprandGenerator_t prng;
    hiprandCreateGenerator(&prng, HIPRAND_RNG_PSEUDO_DEFAULT);

    // Set the seed for the random number generator using the system clock
    hiprandSetPseudoRandomGeneratorSeed(prng, (unsigned long long) clock());

    // Fill the array with random numbers on the device
    hiprandGenerateUniform(prng, A, nr_rows_A * nr_cols_A);
}

// Multiply the arrays A and B on GPU and save the result in C
// C(m,n) = A(m,k) * B(k,n)
void gpu_blas_mmul(const float *A, const float *B, float *C, const int m, const int k, const int n) {
    int lda=m,ldb=k,ldc=m;
    const float alf = 1;
    const float bet = 0;
    const float *alpha = &alf;
    const float *beta = &bet;

    // Create a handle for CUBLAS
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    // Do the actual multiplication
    // HIPBLAS_OP_T means input is row major, HIPBLAS_OP_N means input is column major
//    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, alpha, A, lda, B, ldb, beta, C, ldc);
    hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_T, m, n, k, alpha, A, lda, B, ldb, beta, C, ldc);

    // Destroy the handle
    hipblasDestroy(handle);
}


//Print matrix A(nr_rows_A, nr_cols_A) storage in column-major format
void print_matrix(const float *A, int nr_rows_A, int nr_cols_A) {

    for(int i = 0; i < nr_rows_A; ++i){
        for(int j = 0; j < nr_cols_A; ++j){
            std::cout << A[j * nr_rows_A + i] << " ";
        }
        std::cout << std::endl;
    }
    std::cout << std::endl;
}

K gpu_mm(K A, K B, K C) {
    // Allocate 3 arrays on CPU
    int nr_rows_A, nr_cols_A, nr_rows_B, nr_cols_B, nr_rows_C, nr_cols_C;

    // get shape of input A (assume A/B/C are all square, same shape
    long n=A->n;
    long r=sqrt(n);

    // for simplicity we are going to use square arrays
    nr_rows_A = nr_cols_A = nr_rows_B = nr_cols_B = nr_rows_C = nr_cols_C = r;

    // allocate memory, host arrays
    float *h_A = (float *)malloc(nr_rows_A * nr_cols_A * sizeof(float));
    float *h_B = (float *)malloc(nr_rows_B * nr_cols_B * sizeof(float));
    float *h_C = (float *)malloc(nr_rows_C * nr_cols_C * sizeof(float));

    // Allocate 3 arrays on GPU, device arrays
    float *d_A, *d_B, *d_C;
    float *host_memoryA = (float*) &(kE(A)[0]);
    float *host_memoryB = (float*) &(kE(B)[0]);
    float *host_memoryC = (float*) &(kE(C)[0]);
    size_t size = n * sizeof(float);
    hipMalloc((void **)&d_A, size);
    hipMalloc((void **)&d_B, size);
    hipMalloc(&d_C,nr_rows_C * nr_cols_C * sizeof(float));

    // If you already have useful values in A and B you can copy them in GPU:
    hipMemcpy(d_A, host_memoryA, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, host_memoryB, size, hipMemcpyHostToDevice);

    // Multiply A and B on GPU
    gpu_blas_mmul(d_A, d_B, d_C, nr_rows_A, nr_cols_A, nr_cols_B);

    // Copy the result on host memory
    hipMemcpy(host_memoryC,d_C,nr_rows_C * nr_cols_C * sizeof(float),hipMemcpyDeviceToHost);

    //Free GPU memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);    

    // Free CPU memory
    free(h_A);
    free(h_B);
    free(h_C);

    R r1(C);
}
